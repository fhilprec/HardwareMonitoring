#include "hip/hip_runtime.h"
#include <gflags/gflags.h>
#include <iostream>
#include <cstdint>
#include "hip/hip_runtime_api.h"

#include "storage.hpp"
#include "util.hpp"
#include "dev_util.cuh"

#include "MonitoringInterface.h"


DEFINE_uint32(cuda_device, 0, "Index of CUDA device to use.");
DEFINE_uint32(scale_factor, 5, "Scale factor == size in GB.");
DEFINE_uint32(per_op_repeat, 3, "Repetition of each operation.");
DEFINE_uint32(kernel_ms, 200, "Time to waste in GPU Kernel.");
DEFINE_uint32(store_offset, 0, "Starting offset in file or block device.");
DEFINE_string(ssdpath, "/raid/gds/300G.file", "Path to block device or file.");



#include <hip/hip_runtime.h>

// Define matrix dimensions
#define MATRIX_SIZE 4096

// CUDA kernel for matrix multiplication
__global__ void matrixMultiply(float *A, float *B, float *C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < width && col < width) {
        float sum = 0.0f;
        for (int i = 0; i < width; ++i) {
            sum += A[row * width + i] * B[i * width + col];
        }
        C[row * width + col] = sum;
    }
}

// Function to initialize a matrix with random values
void initializeMatrix(float *matrix, int size) {
    for (int i = 0; i < size * size; ++i) {
        matrix[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

// Replace step 2 with this function
void performGPUWork(int n, uint32_t kernel_ms) {
    const int matrixSize = MATRIX_SIZE * MATRIX_SIZE;
    const int matrixBytes = matrixSize * sizeof(float);

    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    // Allocate host memory
    h_A = new float[matrixSize];
    h_B = new float[matrixSize];
    h_C = new float[matrixSize];

    // Initialize matrices
    initializeMatrix(h_A, MATRIX_SIZE);
    initializeMatrix(h_B, MATRIX_SIZE);

    // Allocate device memory
    hipMalloc(&d_A, matrixBytes);
    hipMalloc(&d_B, matrixBytes);
    hipMalloc(&d_C, matrixBytes);

    // Copy data to device
    hipMemcpy(d_A, h_A, matrixBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matrixBytes, hipMemcpyHostToDevice);

    // Setup execution parameters
    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((MATRIX_SIZE + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (MATRIX_SIZE + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Perform matrix multiplication multiple times
    for (int i = 0; i < n; ++i) {
        matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, MATRIX_SIZE);
        hipDeviceSynchronize();
    }

    // Copy result back to host
    hipMemcpy(h_C, d_C, matrixBytes, hipMemcpyDeviceToHost);

    // Clean up
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main(int argc, char *argv[]){
    gflags::ParseCommandLineFlags(&argc, &argv, true);
    checkCudaErrors(hipSetDevice(FLAGS_cuda_device));
    StorageManager::get().init(FLAGS_ssdpath);

    { // stack for cuda-memcheck
    uint64_t bytes = FLAGS_scale_factor * uint64_t(1<<30);
    uint64_t block_size = bytes>>3;
    util::Log::get().info_fmt("Scale Factor %llu, %.2f GiB, Block Size %llu, %.2f GiB",
                                bytes, bytes / double(1<<30),
                                block_size, block_size / double(1<<30));
    char *hst_ptr,*dev_ptr;
    checkCudaErrors(hipHostMalloc(&hst_ptr,bytes));
    checkCudaErrors(hipMalloc(&dev_ptr,bytes));
    if (uint64_t(dev_ptr) % 4096 != 0){
        util::Log::get().info_fmt("dev ptr %p", dev_ptr);
    }
    if (uint64_t(hst_ptr) % 4096 != 0){
        util::Log::get().info_fmt("hst ptr %p", hst_ptr);
    }
    checkCuFileError(cuFileBufRegister(dev_ptr, bytes, 0));

    util::Timer total_timer;
    util::Timer timer;
    start_monitoring();
    // 1) Read from storage
    for (int i = 0; i < FLAGS_per_op_repeat; ++i){
        util::ThreadPool::parallel_n(8, [&](int tid) {
            // every thread one read for now
            auto ret = cuFileRead(StorageManager::get().cfh, dev_ptr, block_size, FLAGS_store_offset + tid*block_size,
                                    tid*block_size);
            if (ret != block_size) util::Log::get().info_fmt("Tried reading %llu bytes, but read %llu", block_size, ret);
        });
    }
    util::Log::get().info_fmt("Storage reads took %.2f ms", timer.elapsed());
    timer.reset();
    int n = 5;

    // 2) Waste some time on GPU
    for (int i = 0; i < n; ++i){
        // util::waiting_kernel<<<1,1>>>(FLAGS_kernel_ms * 0);
        performGPUWork(n, FLAGS_kernel_ms * 1);
        checkCudaErrors(hipDeviceSynchronize());
    }
    util::Log::get().info_fmt("Kernels took %.2f ms", timer.elapsed());
    timer.reset();
        // std::this_thread::sleep_for(std::chrono::milliseconds(5000));
    std::this_thread::sleep_for(std::chrono::milliseconds(500));


    // 3) Copy to main memory
    for (int i = 0; i < n; ++i){
        checkCudaErrors(hipMemcpy(hst_ptr, dev_ptr, bytes, hipMemcpyDefault));
    }
    util::Log::get().info_fmt("Memcpys took %.2f ms", timer.elapsed());
    timer.reset();

    // 4) Also waste some time on CPU
    for (int i = 0; i < n; ++i){
        util::ThreadPool::parallel_n(8, [&](int tid) {
            util::Timer::sleep_ms(200);
            int sum = 0;
            for(int i  = 0; i < 10000000; i++){
                sum += i * i * i * i;
            }
        });
    }
    util::Log::get().info_fmt("CPU threads took %.2f ms", timer.elapsed());
    timer.reset();

    // 5) Write to storage
    for (int i = 0; i < FLAGS_per_op_repeat; ++i){
        StorageManager::get().host_write_bytes(hst_ptr, bytes, FLAGS_store_offset);
    }
    util::Log::get().info_fmt("Storage writes took %.2f ms", timer.elapsed());
    util::Log::get().info_fmt("Total took %.2f ms", total_timer.elapsed());

    stop_monitoring();
    // clean up
    checkCuFileError(cuFileBufDeregister(dev_ptr));
    checkCudaErrors(hipHostFree(hst_ptr));
    checkCudaErrors(hipFree(dev_ptr));
    } // stack for cuda-memcheck


    checkCudaErrors(hipDeviceReset());
}