#include <gflags/gflags.h>
#include <iostream>
#include <cstdint>
#include "hip/hip_runtime_api.h"

#include "storage.hpp"
#include "util.hpp"
#include "dev_util.cuh"


DEFINE_uint32(cuda_device, 0, "Index of CUDA device to use.");
DEFINE_uint32(scale_factor, 5, "Scale factor == size in GB.");
DEFINE_uint32(per_op_repeat, 3, "Repetition of each operation.");
DEFINE_uint32(kernel_ms, 200, "Time to waste in GPU Kernel.");
DEFINE_uint32(store_offset, 0, "Starting offset in file or block device.");
DEFINE_string(ssdpath, "/raid/gds/300G.file", "Path to block device or file.");

int main(int argc, char *argv[]){
    gflags::ParseCommandLineFlags(&argc, &argv, true);
    checkCudaErrors(hipSetDevice(FLAGS_cuda_device));
    StorageManager::get().init(FLAGS_ssdpath);

    { // stack for cuda-memcheck
    uint64_t bytes = FLAGS_scale_factor * uint64_t(1<<30);
    uint64_t block_size = bytes>>3;
    util::Log::get().info_fmt("Scale Factor %llu, %.2f GiB, Block Size %llu, %.2f GiB",
                                bytes, bytes / double(1<<30),
                                block_size, block_size / double(1<<30));
    char *hst_ptr,*dev_ptr;
    checkCudaErrors(hipHostMalloc(&hst_ptr,bytes));
    checkCudaErrors(hipMalloc(&dev_ptr,bytes));
    if (uint64_t(dev_ptr) % 4096 != 0){
        util::Log::get().info_fmt("dev ptr %p", dev_ptr);
    }
    if (uint64_t(hst_ptr) % 4096 != 0){
        util::Log::get().info_fmt("hst ptr %p", hst_ptr);
    }
    checkCuFileError(cuFileBufRegister(dev_ptr, bytes, 0));

    util::Timer total_timer;
    util::Timer timer;

    // 1) Read from storage
    for (int i = 0; i < FLAGS_per_op_repeat; ++i){
        util::ThreadPool::parallel_n(8, [&](int tid) {
            // every thread one read for now
            auto ret = cuFileRead(StorageManager::get().cfh, dev_ptr, block_size, FLAGS_store_offset + tid*block_size,
                                    tid*block_size);
            if (ret != block_size) util::Log::get().info_fmt("Tried reading %llu bytes, but read %llu", block_size, ret);
        });
    }
    util::Log::get().info_fmt("Storage reads took %.2f ms", timer.elapsed());
    timer.reset();

    // 2) Waste some time on GPU
    for (int i = 0; i < FLAGS_per_op_repeat; ++i){
        util::waiting_kernel<<<1,1>>>(FLAGS_kernel_ms * 1000);
        checkCudaErrors(hipDeviceSynchronize());
    }
    util::Log::get().info_fmt("Kernels took %.2f ms", timer.elapsed());
    timer.reset();

    // 3) Copy to main memory
    for (int i = 0; i < FLAGS_per_op_repeat; ++i){
        checkCudaErrors(hipMemcpy(hst_ptr, dev_ptr, bytes, hipMemcpyDefault));
    }
    util::Log::get().info_fmt("Memcpys took %.2f ms", timer.elapsed());
    timer.reset();

    // 4) Also waste some time on CPU
    for (int i = 0; i < FLAGS_per_op_repeat; ++i){
        util::ThreadPool::parallel_n(8, [&](int tid) {
            util::Timer::sleep_ms(200);
        });
    }
    util::Log::get().info_fmt("CPU threads took %.2f ms", timer.elapsed());
    timer.reset();

    // 5) Write to storage
    for (int i = 0; i < FLAGS_per_op_repeat; ++i){
        StorageManager::get().host_write_bytes(hst_ptr, bytes, FLAGS_store_offset);
    }
    util::Log::get().info_fmt("Storage writes took %.2f ms", timer.elapsed());
    util::Log::get().info_fmt("Total took %.2f ms", total_timer.elapsed());


    // clean up
    checkCuFileError(cuFileBufDeregister(dev_ptr));
    checkCudaErrors(hipHostFree(hst_ptr));
    checkCudaErrors(hipFree(dev_ptr));
    } // stack for cuda-memcheck


    checkCudaErrors(hipDeviceReset());
}