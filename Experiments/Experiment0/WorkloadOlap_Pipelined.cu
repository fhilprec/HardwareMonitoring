#include "hip/hip_runtime.h"
#include <gflags/gflags.h>
#include <iostream>
#include <cstdint>
#include "hip/hip_runtime_api.h"

#include "storage.hpp"
#include "util.hpp"
#include "dev_util.cuh"

#include "MonitoringInterface.h"


DEFINE_uint32(cuda_device, 0, "Index of CUDA device to use.");
DEFINE_uint32(scale_factor, 5, "Scale factor == size in GB.");
DEFINE_uint32(per_op_repeat, 3, "Repetition of each operation.");
DEFINE_uint32(kernel_ms, 200, "Time to waste in GPU Kernel.");
DEFINE_uint32(store_offset, 0, "Starting offset in file or block device.");
DEFINE_string(ssdpath, "/raid/gds/300G.file", "Path to block device or file.");



#include <hip/hip_runtime.h>

// Define matrix dimensions
#define MATRIX_SIZE 4096
#define NUM_STREAMS 4

// CUDA kernel for matrix multiplication
__global__ void matrixMultiply(float *A, float *B, float *C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < width && col < width) {
        float sum = 0.0f;
        for (int i = 0; i < width; ++i) {
            sum += A[row * width + i] * B[i * width + col];
        }
        C[row * width + col] = sum;
    }
}

// Function to initialize a matrix with random values
void initializeMatrix(float *matrix, int size) {
    for (int i = 0; i < size * size; ++i) {
        matrix[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

// Replace step 2 with this function
void performGPUWork(int n, uint32_t kernel_ms) {
    const int matrixSize = MATRIX_SIZE * MATRIX_SIZE;
    const int matrixBytes = matrixSize * sizeof(float);

    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    // Allocate host memory
    h_A = new float[matrixSize];
    h_B = new float[matrixSize];
    h_C = new float[matrixSize];

    // Initialize matrices
    initializeMatrix(h_A, MATRIX_SIZE);
    initializeMatrix(h_B, MATRIX_SIZE);

    // Allocate device memory
    hipMalloc(&d_A, matrixBytes);
    hipMalloc(&d_B, matrixBytes);
    hipMalloc(&d_C, matrixBytes);

    // Copy data to device
    hipMemcpy(d_A, h_A, matrixBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matrixBytes, hipMemcpyHostToDevice);

    // Setup execution parameters
    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((MATRIX_SIZE + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (MATRIX_SIZE + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Perform matrix multiplication multiple times
    for (int i = 0; i < n; ++i) {
        matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, MATRIX_SIZE);
        hipDeviceSynchronize();
    }

    // Copy result back to host
    hipMemcpy(h_C, d_C, matrixBytes, hipMemcpyDeviceToHost);

    // Clean up
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void performGPUWork(float *d_A, float *d_B, float *d_C, int n, hipStream_t stream) {
    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((MATRIX_SIZE + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (MATRIX_SIZE + threadsPerBlock.y - 1) / threadsPerBlock.y);

    for (int i = 0; i < n; ++i) {
        matrixMultiply<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_A, d_B, d_C, MATRIX_SIZE);
    }
}

void performCPUWork(int tid) {
    util::Timer::sleep_ms(200);
    int sum = 0;
    for(int i = 0; i < 10000000; i++){
        sum += i * i * i * i;
    }
}


int main(int argc, char *argv[]) {
    gflags::ParseCommandLineFlags(&argc, &argv, true);
    checkCudaErrors(hipSetDevice(FLAGS_cuda_device));
    StorageManager::get().init(FLAGS_ssdpath);

    uint64_t bytes = FLAGS_scale_factor * uint64_t(1<<30);
    uint64_t chunk_size = bytes / NUM_STREAMS;
    util::Log::get().info_fmt("Scale Factor %llu, %.2f GiB, Chunk Size %llu, %.2f GiB",
                              bytes, bytes / double(1<<30),
                              chunk_size, chunk_size / double(1<<30));

    std::vector<char*> hst_ptrs(NUM_STREAMS);
    std::vector<char*> dev_ptrs(NUM_STREAMS);
    std::vector<hipStream_t> streams(NUM_STREAMS);
    std::vector<float*> d_A(NUM_STREAMS), d_B(NUM_STREAMS), d_C(NUM_STREAMS);

    for (int i = 0; i < NUM_STREAMS; ++i) {
        checkCudaErrors(hipHostMalloc(&hst_ptrs[i], chunk_size));
        checkCudaErrors(hipMalloc(&dev_ptrs[i], chunk_size));
        checkCudaErrors(hipStreamCreate(&streams[i]));
        checkCuFileError(cuFileBufRegister(dev_ptrs[i], chunk_size, 0));

        // Allocate memory for matrix multiplication
        checkCudaErrors(hipMalloc(&d_A[i], MATRIX_SIZE * MATRIX_SIZE * sizeof(float)));
        checkCudaErrors(hipMalloc(&d_B[i], MATRIX_SIZE * MATRIX_SIZE * sizeof(float)));
        checkCudaErrors(hipMalloc(&d_C[i], MATRIX_SIZE * MATRIX_SIZE * sizeof(float)));
    }

    util::Timer total_timer;
    start_monitoring();

    for (int repeat = 0; repeat < FLAGS_per_op_repeat; ++repeat) {
        for (int i = 0; i < NUM_STREAMS; ++i) {
            uint64_t offset = FLAGS_store_offset + i * chunk_size;
            
            // 1. Read from storage (asynchronous)
            auto ret = cuFileRead(StorageManager::get().cfh, dev_ptrs[i], chunk_size, offset, 0);
            if (ret != chunk_size) util::Log::get().info_fmt("Tried reading %llu bytes, but read %llu", chunk_size, ret);

            // 2. Perform GPU work (asynchronous)
            performGPUWork(d_A[i], d_B[i], d_C[i], 5, streams[i]);

            // 3. Copy to main memory (asynchronous)
            checkCudaErrors(hipMemcpyAsync(hst_ptrs[i], dev_ptrs[i], chunk_size, hipMemcpyDeviceToHost, streams[i]));

            // 4. Perform CPU work (asynchronous)
            std::thread cpu_thread(performCPUWork, i);
            cpu_thread.detach();  // Allow the thread to run independently

            // 5. Write to storage (will be synchronized before the next iteration)
            hipStreamSynchronize(streams[i]);  // Ensure all operations are complete before writing
            StorageManager::get().host_write_bytes(hst_ptrs[i], chunk_size, offset);
        }
    }

    // Wait for all streams to complete
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipStreamSynchronize(streams[i]);
    }

    util::Log::get().info_fmt("Total took %.2f ms", total_timer.elapsed());

    stop_monitoring();

    // Clean up
    for (int i = 0; i < NUM_STREAMS; ++i) {
        checkCuFileError(cuFileBufDeregister(dev_ptrs[i]));
        checkCudaErrors(hipHostFree(hst_ptrs[i]));
        checkCudaErrors(hipFree(dev_ptrs[i]));
        checkCudaErrors(hipFree(d_A[i]));
        checkCudaErrors(hipFree(d_B[i]));
        checkCudaErrors(hipFree(d_C[i]));
        checkCudaErrors(hipStreamDestroy(streams[i]));
    }

    checkCudaErrors(hipDeviceReset());
    return 0;
}