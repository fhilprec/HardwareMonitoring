#include <gflags/gflags.h>
#include <iostream>
#include <cstdint>
#include "hip/hip_runtime_api.h"

#include "storage.hpp"
#include "util.hpp"
#include "dev_util.cuh"

#include "MonitoringInterface.h"

DEFINE_uint32(cuda_device, 0, "Index of CUDA device to use.");
DEFINE_uint32(scale_factor, 8, "Scale factor == size in GB.");
DEFINE_uint32(kernel_ms, 200, "Time to waste in GPU Kernel.");
DEFINE_uint32(store_offset, 0, "Starting offset in file or block device.");
DEFINE_string(ssdpath, "/raid/gds/300G.file", "Path to block device or file.");

int main(int argc, char *argv[]){
    start_monitoring();
    gflags::ParseCommandLineFlags(&argc, &argv, true);
    checkCudaErrors(hipSetDevice(FLAGS_cuda_device));
    StorageManager::get().init(FLAGS_ssdpath);
    const int THREAD_NUM = 8;

    { // stack for cuda-memcheck
    uint64_t bytes = FLAGS_scale_factor * uint64_t(1<<30);
    uint64_t block_size = bytes/THREAD_NUM;
    util::Log::get().info_fmt("Scale Factor %llu, %.2f GiB, Block Size %llu, %.2f GiB", bytes, bytes / double(1<<30),
                                block_size, block_size / double(1<<30));
    char *hst_ptr,*dev_ptr;
    hipStream_t streams[THREAD_NUM];
    for (int i = 0; i<THREAD_NUM; ++i) checkCudaErrors(hipStreamCreate(streams+i));

    checkCudaErrors(hipHostMalloc(&hst_ptr,bytes));
    checkCudaErrors(hipMalloc(&dev_ptr,bytes));
    if (uint64_t(dev_ptr) % (1<<12) != 0){
        util::Log::get().info_fmt("dev ptr %p", dev_ptr);
    }
    if (uint64_t(hst_ptr) % (1<<12) != 0){
        util::Log::get().info_fmt("hst ptr %p", hst_ptr);
    }
    checkCuFileError(cuFileBufRegister(dev_ptr, bytes, 0));

    util::Timer timer;

    util::ThreadPool::parallel_n(THREAD_NUM, [&](int tid) {
        checkCudaErrors(hipSetDevice(FLAGS_cuda_device));
        // 1) read block
        auto ret = cuFileRead(StorageManager::get().cfh, dev_ptr, block_size, FLAGS_store_offset + tid*block_size, tid*block_size);
        if (ret != block_size) util::Log::get().info_fmt("Tried reading %llu bytes, but read %llu", block_size, ret);
        // 2) Waste some time on GPU
        util::waiting_kernel<<<1,1,0,streams[tid]>>>(FLAGS_kernel_ms * 1000);
        // 3) Copy to main memory
        checkCudaErrors(hipMemcpyAsync(hst_ptr, dev_ptr, bytes, hipMemcpyDefault, streams[tid]));
        // 4) Also waste some CPU and Mem resources
        // Note: hipMemcpyHostToHost usually not what you want to do
        checkCudaErrors(hipMemcpyAsync(hst_ptr + tid*block_size, hst_ptr + tid*block_size + (block_size>>1), (block_size>>1), hipMemcpyDefault, streams[tid]));
        // 5) Write to storage
        checkCudaErrors(hipStreamSynchronize(streams[tid]));
        StorageManager::get().host_write_bytes(hst_ptr + tid*block_size, block_size, FLAGS_store_offset + tid*block_size);
    });

    stop_monitoring();
    util::Log::get().info_fmt("Total took %.2f ms", timer.elapsed());

    // clean up
    for (int i = 0; i<THREAD_NUM; ++i) checkCudaErrors(hipStreamDestroy(streams[i]));
    checkCuFileError(cuFileBufDeregister(dev_ptr));
   
    checkCudaErrors(hipHostFree(hst_ptr));
    checkCudaErrors(hipFree(dev_ptr));
    } // stack for cuda-memcheck
    checkCudaErrors(hipDeviceReset());
}
